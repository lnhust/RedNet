#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"
#include "parser.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "rnn_layer.h"
#include "gru_layer.h"
#include "crnn_layer.h"
#include "detection_layer.h"
#include "region_layer.h"
#include "convolutional_layer.h"
#include "activation_layer.h"
#include "maxpool_layer.h"
#include "reorg_layer.h"
#include "avgpool_layer.h"
#include "normalization_layer.h"
#include "batchnorm_layer.h"
#include "cost_layer.h"
#include "local_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
#include "route_layer.h"
#include "shortcut_layer.h"
#include "blas.h"
}

float * get_network_output_gpu_layer(network net, int i);
float * get_network_delta_gpu_layer(network net, int i);
float * get_network_output_gpu(network net);
float * get_network_output_layer_gpu(network net, int i);

void forward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    for(i = 0; i < net.n; ++i){
        state.index = i;
        layer l = net.layers[i];
        if(l.delta_gpu){
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }
        l.forward_gpu(l, state);
        state.input = l.output_gpu;
    }
}

void backward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    float * original_input = state.input;
    float * original_delta = state.delta;
    for(i = net.n-1; i >= 0; --i){
        state.index = i;
        layer l = net.layers[i];
        if(i == 0){
            state.input = original_input;
            state.delta = original_delta;
        }else{
            layer prev = net.layers[i-1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
        }
        l.backward_gpu(l, state);
    }
}

void update_network_gpu(network net)
{
    int i;
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    for(i = 0; i < net.n; ++i){
        layer l = net.layers[i];
        l.t = get_current_batch(net);
        if(l.update_gpu){
            l.update_gpu(l, update_batch, rate, net.momentum, net.decay);
        }
    }
}


void forward_network_gpu_use_flag(network net, network_state state, int* flag, int isTrain)
{
    state.workspace = net.workspace;
    int i;
    for(i = 0; i < net.n; ++i){
        state.index = i;
        layer l = net.layers[i];
        if(l.delta_gpu){
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }
        l.forward_gpu(l, state);
        state.input = l.output_gpu;
        
        flag[i] = 1;
        if (l.type == COST)
        {
//        	float* out = (float*)calloc(net.layers[i - 1].outputs*net.layers[i - 1].batch, sizeof(float));
//          cuda_copy_array(net.layers[i - 1].output_gpu, out, net.layers[i - 1].outputs*net.layers[i - 1].batch);
        	if (net.early_stop)
        	{
        		//get upper threashold
				float upper = net.upperbound;
				if(isTrain)
				{
					float precentage = (float)(*net.seen) / net.N / 50;
					float prob_rand = 1.0 / net.nclasses;
		//        	printf("%d,%d,%f", *net.seen, net.N, precentage);				
					upper = (net.upperbound - prob_rand) * precentage + prob_rand;
					upper = upper > net.upperbound ? net.upperbound : upper;
				}
				else
				
				//if train use voting to deciside whether to stop
				//else use one sample.
				float* out = get_network_output_layer_gpu(net, i - 1);
				int outputs = net.layers[i - 1].outputs;
				int batch_size = net.batch;
				int indexes;
				
				int b;
				int early_stop_number = 0;
				float mean_prob = 0;
				for (b = 0; b < batch_size; b++)
				{
					top_k(out + outputs * b, outputs, 1, &indexes);
					if(out[indexes + outputs * b, outputs] >= upper)
					{
						early_stop_number++;
						mean_prob += out[indexes + outputs * b, outputs];
					}
				}
				
				if (net.print2console)
					if (batch_size == 1)
						printf("Cost layer AT %d with probability %.6f of type %d and threshold: %.6f", i, out[indexes], indexes, upper);
					else
						printf("Cost layer AT %d higher than threshold: %.6f with mean probability %.6f of %d samples", 
								i, upper, mean_prob / batch_size, early_stop_number);
						
				if(early_stop_number >= batch_size / 2)
				{
					if (net.print2console)
						printf("----------------------------STOP!\n");
					break;
				}
        	}
			if (i != net.n - 1)
			{
				if (net.early_stop && net.print2console)
					printf("----------------------------DOESN'T STOP!\n");
				int i_forward = i;
				//Cost layer set to be false
				flag[i_forward--] = 0;
				while(net.layers[i_forward].type != CONVOLUTIONAL)
					flag[i_forward--] = 0;
				//last fully convolutional layer set to be false
				flag[i_forward--] = 0;
				state.input = net.layers[i_forward].output_gpu;
			}
			else
			{
				if (net.early_stop && net.print2console)
					printf("----------------------------STOP!\n");
			}
        }
    }    

	if (net.early_stop && net.print2console)
	{
		printf("layer");
		int total_ignored = 0;
		for (i = 0; i < net.n; i++)
			if (!flag[i])
			{
				printf(" %d", i);
				total_ignored++;
			}
		if (total_ignored)
			printf(" is ignored!\n");
		else
			printf("None is ignored!\n");
	}
}

void backward_network_gpu_use_flag(network net, network_state state, int* flag)
{
    state.workspace = net.workspace;
    int i;
    float * original_input = state.input;
    float * original_delta = state.delta;
    int last_layer, first_layer;
    for (i = net.n - 1; i >= 0; i--)
    	if (flag[i]) break;
    last_layer = i;
    
    for(i = last_layer; i > 0; i--)
    	if (!flag[i - 1]) break;
    first_layer = i;
    
    printf("Backward and Update layer:");
    for(i = last_layer; i >= first_layer; --i){    
    	printf(" %d", i);
    	
        state.index = i;
        layer l = net.layers[i];
        if(i == 0){
            state.input = original_input;
            state.delta = original_delta;
        }else{
            layer prev = net.layers[i-1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
        }
        l.backward_gpu(l, state);
    }
    printf("\n");
}

void update_network_gpu_use_flag(network net, int* flag)
{
    int i;
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    
    int last_layer, first_layer;
    for (i = net.n - 1; i >= 0; i--)
    	if (flag[i]) break;
    last_layer = i;
    
    for(i = last_layer; i > 0; i--)
    	if (!flag[i - 1]) break;
    first_layer = i;
    
    for(i = first_layer; i < last_layer; ++i){    		
        layer l = net.layers[i];
        l.t = get_current_batch(net);
        if(l.update_gpu){
            l.update_gpu(l, update_batch, rate, net.momentum, net.decay);
        }
    }
}

void forward_backward_network_gpu(network net, float *x, float *y)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(net.layers[net.n-1].truths) y_size = net.layers[net.n-1].truths*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    state.truth = *net.truth_gpu;
    state.train = 1;
    forward_network_gpu(net, state);
    backward_network_gpu(net, state);
}

void forward_backward_network_gpu_use_flag(network net, float *x, float *y, int* flag)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(net.layers[net.n-1].truths) y_size = net.layers[net.n-1].truths*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    state.truth = *net.truth_gpu;
    state.train = 1;
    forward_network_gpu_use_flag(net, state, flag, 1);
    backward_network_gpu_use_flag(net, state, flag);
}


float train_network_datum_gpu(network net, float *x, float *y)
{
	int* flag = (int*)calloc(net.n, sizeof(int));
    *net.seen += net.batch;
    forward_backward_network_gpu_use_flag(net, x, y, flag);
    float error = get_network_cost(net);
    if (((*net.seen) / net.batch) % net.subdivisions == 0) update_network_gpu_use_flag(net, flag);
	
	free(flag);
    return error;
}

typedef struct {
    network net;
    data d;
    float *err;
} train_args;

void *train_thread(void *ptr)
{
    train_args args = *(train_args*)ptr;
    free(ptr);
    cuda_set_device(args.net.gpu_index);
    *args.err = train_network(args.net, args.d);
    return 0;
}

pthread_t train_network_in_thread(network net, data d, float *err)
{
    pthread_t thread;
    train_args *ptr = (train_args *)calloc(1, sizeof(train_args));
    ptr->net = net;
    ptr->d = d;
    ptr->err = err;
    if(pthread_create(&thread, 0, train_thread, ptr)) error("Thread creation failed");
    return thread;
}

void pull_updates(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.n*l.size*l.size*l.c);
        if(l.scale_updates) cuda_pull_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
}

void push_updates(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.n*l.size*l.size*l.c);
        if(l.scale_updates) cuda_push_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
}

void update_layer(layer l, network net)
{
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    l.t = get_current_batch(net);
    if(l.update_gpu){
        l.update_gpu(l, update_batch, rate, net.momentum, net.decay);
    }
}

void merge_weights(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.n*l.size*l.size*l.c, 1, l.weights, 1, base.weights, 1);
        if (l.scales) {
            axpy_cpu(l.n, 1, l.scales, 1, base.scales, 1);
        }
    } else if(l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.outputs*l.inputs, 1, l.weights, 1, base.weights, 1);
    }
}

void scale_weights(layer l, float s)
{
    if (l.type == CONVOLUTIONAL) {
        scal_cpu(l.n, s, l.biases, 1);
        scal_cpu(l.n*l.size*l.size*l.c, s, l.weights, 1);
        if (l.scales) {
            scal_cpu(l.n, s, l.scales, 1);
        }
    } else if(l.type == CONNECTED) {
        scal_cpu(l.outputs, s, l.biases, 1);
        scal_cpu(l.outputs*l.inputs, s, l.weights, 1);
    }
}


void pull_weights(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_pull_array(l.biases_gpu, l.biases, l.n);
        cuda_pull_array(l.weights_gpu, l.weights, l.n*l.size*l.size*l.c);
        if(l.scales) cuda_pull_array(l.scales_gpu, l.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_pull_array(l.biases_gpu, l.biases, l.outputs);
        cuda_pull_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
    }
}

void push_weights(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.biases_gpu, l.biases, l.n);
        cuda_push_array(l.weights_gpu, l.weights, l.n*l.size*l.size*l.c);
        if(l.scales) cuda_push_array(l.scales_gpu, l.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.biases_gpu, l.biases, l.outputs);
        cuda_push_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
    }
}

void distribute_weights(layer l, layer base)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.biases_gpu, base.biases, l.n);
        cuda_push_array(l.weights_gpu, base.weights, l.n*l.size*l.size*l.c);
        if(base.scales) cuda_push_array(l.scales_gpu, base.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.biases_gpu, base.biases, l.outputs);
        cuda_push_array(l.weights_gpu, base.weights, l.outputs*l.inputs);
    }
}


void merge_updates(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.n*l.size*l.size*l.c, 1, l.weight_updates, 1, base.weight_updates, 1);
        if (l.scale_updates) {
            axpy_cpu(l.n, 1, l.scale_updates, 1, base.scale_updates, 1);
        }
    } else if(l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.outputs*l.inputs, 1, l.weight_updates, 1, base.weight_updates, 1);
    }
}

void distribute_updates(layer l, layer base)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.n*l.size*l.size*l.c);
        if(base.scale_updates) cuda_push_array(l.scale_updates_gpu, base.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.outputs*l.inputs);
    }
}

void sync_layer(network *nets, int n, int j)
{
    //printf("Syncing layer %d\n", j);
    int i;
    network net = nets[0];
    layer base = net.layers[j];
    cuda_set_device(net.gpu_index);
    pull_weights(base);
    for (i = 1; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        pull_weights(l);
        merge_weights(l, base);
    }
    scale_weights(base, 1./n);
    for (i = 0; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        distribute_weights(l, base);
    }
    //printf("Done syncing layer %d\n", j);
}

typedef struct{
    network *nets;
    int n;
    int j;
} sync_args;

void *sync_layer_thread(void *ptr)
{
    sync_args args = *(sync_args*)ptr;
    sync_layer(args.nets, args.n, args.j);
    free(ptr);
    return 0;
}

pthread_t sync_layer_in_thread(network *nets, int n, int j)
{
    pthread_t thread;
    sync_args *ptr = (sync_args *)calloc(1, sizeof(sync_args));
    ptr->nets = nets;
    ptr->n = n;
    ptr->j = j;
    if(pthread_create(&thread, 0, sync_layer_thread, ptr)) error("Thread creation failed");
    return thread;
}

void sync_nets(network *nets, int n, int interval)
{
    int j;
    int layers = nets[0].n;
    pthread_t *threads = (pthread_t *) calloc(layers, sizeof(pthread_t));

    *nets[0].seen += interval * (n-1) * nets[0].batch * nets[0].subdivisions;
    for (j = 0; j < n; ++j){
        *nets[j].seen = *nets[0].seen;
    }
    for (j = 0; j < layers; ++j) {
        threads[j] = sync_layer_in_thread(nets, n, j);
    }
    for (j = 0; j < layers; ++j) {
        pthread_join(threads[j], 0);
    }
    free(threads);
}

float train_networks(network *nets, int n, data d, int interval)
{
    int i;
    int batch = nets[0].batch;
    int subdivisions = nets[0].subdivisions;
    assert(batch * subdivisions * n == d.X.rows);
    pthread_t *threads = (pthread_t *) calloc(n, sizeof(pthread_t));
    float *errors = (float *) calloc(n, sizeof(float));

    float sum = 0;
    for(i = 0; i < n; ++i){
        data p = get_data_part(d, i, n);
        threads[i] = train_network_in_thread(nets[i], p, errors + i);
    }
    for(i = 0; i < n; ++i){
        pthread_join(threads[i], 0);
        //printf("%f\n", errors[i]);
        sum += errors[i];
    }
    //hipDeviceSynchronize();
    if (get_current_batch(nets[0]) % interval == 0) {
        printf("Syncing... ");
        fflush(stdout);
        sync_nets(nets, n, interval);
        printf("Done!\n");
    }
    //hipDeviceSynchronize();
    free(threads);
    free(errors);
    return (float)sum/(n);
}

float *get_network_output_layer_gpu(network net, int i)
{
    layer l = net.layers[i];
    cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);
    return l.output;
}

float *get_network_output_gpu(network net)
{
    int i;
    for(i = net.n-1; i > 0; --i) if(net.layers[i].type != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float *get_network_output_gpu_from_index(network net, int index)
{
    int i;
    for(i = index; i > 0; --i) if(net.layers[i].type != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(network net, float *input)
{
    cuda_set_device(net.gpu_index);
    int size = get_network_input_size(net) * net.batch;
    int i, lindex = 0;
    network_state state;
    state.index = 0;
    state.net = net;
    state.input = cuda_make_array(input, size);
    state.truth = 0;
    state.train = 0;
    state.delta = 0;
    int* flag = (int*)calloc(net.n, sizeof(int));
    forward_network_gpu_use_flag(net, state, flag, 0);
    for (i = 0; i < net.n; i++)
    	if (flag[i]) lindex = i;
    float *out = get_network_output_gpu_from_index(net, lindex - 1);
    cuda_free(state.input);
    free(flag);
    return out;
}

